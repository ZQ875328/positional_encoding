#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <bit>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAGuard.h>

template<typename T>
__global__ void pe_kernel(const T* data, const uint32_t data_len, const uint64_t* vec, const uint32_t vec_len, float* out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr float shift_f = std::bit_cast<float>(0x30000000);
    const uint32_t out_len = data_len * vec_len * 2;

    __shared__ T _data[32];
    __shared__ uint64_t _vec[256];
    for (int i = threadIdx.x; i < vec_len; i+=32){
        _vec[i] = vec[i];
    }

    _data[threadIdx.x] = data[idx];

    uint32_t vec_pos = threadIdx.x;
    uint32_t data_pos = 0;

    uint32_t out_pos = blockIdx.x * blockDim.x * vec_len * 2 + threadIdx.x;

    __shared__ float o[2][33];

    for(int i = 0; i < vec_len; i++){
        uint32_t enc = (_data[data_pos] * _vec[vec_pos]) >> 32;
        int32_t code = std::bit_cast<int32_t>(enc);

        float code_f = code * shift_f;

        float s;
        float c;
        sincospif(code_f, &s, &c);
        o[0][threadIdx.x] = s;
        o[1][threadIdx.x] = c;

        float o1 = o[threadIdx.x % 2][threadIdx.x / 2];
        float o2 = o[threadIdx.x % 2][threadIdx.x / 2 + 16];
        if (out_pos < out_len){
            out[out_pos] = o1;
        }
        out_pos += 32;
        if (out_pos < out_len){
            out[out_pos] = o2;
        }
        out_pos += 32;
        vec_pos += 32;
        if (vec_pos >= vec_len){
            vec_pos -= vec_len;
            ++data_pos;
        }
    }
}

torch::Tensor pe(torch::Tensor input, torch::Tensor vec) {
    constexpr uint32_t width = 64;
    constexpr uint32_t threads = 32;
    auto options = torch::TensorOptions()
        .dtype(torch::kFloat32)
        .layout(torch::kStrided)
        .device(torch::kCUDA, input.device().index())
        .requires_grad(false);
    
    auto out_size = input.sizes().vec();
    out_size.push_back(vec.size(0) * 2);
    const auto blocks = (torch::numel(input) + threads - 1) / threads;

    auto result = torch::empty(torch::IntArrayRef({out_size}), options);

    dim3 threads_per_block(threads);
    dim3 number_of_blocks(blocks);

    auto type = input.scalar_type();

    if (type == torch::kUInt64){
        pe_kernel<<<number_of_blocks, threads_per_block>>>(
            input.data_ptr<uint64_t>(), torch::numel(input), vec.data_ptr<uint64_t>(), torch::numel(vec), result.data_ptr<float>()
        );
    } else if (type == torch::kUInt32)
    {
        pe_kernel<<<number_of_blocks, threads_per_block>>>(
            input.data_ptr<uint32_t>(), torch::numel(input), vec.data_ptr<uint64_t>(), torch::numel(vec), result.data_ptr<float>()
        );
    } else {
        throw std::runtime_error("Unsupported type");
    }
    
    return result;
}